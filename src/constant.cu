#include "hip/hip_runtime.h"
#include "constant.cuh"

const double TOLERANCE = 1e-10;

ConstantApprox::ConstantApprox(CImg<unsigned char> *img, double step, double ds_) : stepSize(step), ds(ds_) {
	// create pixel array representation
	maxX = img->width();
	maxY = img->height();
	cout << "image is " << maxX << "x" << maxY << endl;
	// allocate shared space for pixel array
	hipMallocManaged(&pixArr, maxX * maxY * sizeof(Pixel));
	bool isGrayscale = (img->spectrum() == 1);
	for(int x = 0; x < maxX; x++) {
		for(int y = 0; y < maxY; y++) {
			int ind = x * maxY + y; // 1D pixel index
			if(isGrayscale) {
				pixArr[ind] = Pixel(x, y, (*img)(x, y));
			} else {
				int rgb[3];
				for(int i = 0; i < 3; i++) {
					rgb[i] = (*img)(x, y, 0, i);
				}
				int r = (*img)(x, y, 0, 0);
				pixArr[ind] = Pixel(x, y, rgb[0], rgb[1], rgb[2]);
			}
		}
	}
}

void ConstantApprox::initialize(vector<Point> *pts, vector<array<int, 3>> &inds) {
	// load in points of triangulation
	numPoints = pts->size();
	// allocate shared space for points
	hipMallocManaged(&points, numPoints * sizeof(Point));
	// copy everything in TODO: make this more efficient (get directly from source)
	for(int i = 0; i < numPoints; i++) {
		points[i] = pts->at(i);
	}

	// now load in all the triangles
	numTri = inds.size();
	// allocate shared space for triangles and colors
	hipMallocManaged(&triArr, numTri * sizeof(Triangle));
	hipMallocManaged(&grays, numTri * sizeof(double));
	/*
	hipMallocManaged(&reds, numTri * sizeof(double));
	hipMallocManaged(&greens, numTri * sizeof(double));
	hipMallocManaged(&blues, numTri * sizeof(double));
	*/

	double maxLength = 0; // get maximum side length of a triangle for space allocation
	faces = inds;
	for(int i = 0; i < numTri; i++) {
		array<int, 3> t = inds.at(i); // vertex indices for this triangle
		// constructor takes point addresses
		triArr[i] = Triangle(points + t.at(0), points + t.at(1), points + t.at(2));
		maxLength = max(maxLength, triArr[i].maxLength());
	}
	imageInt = new double[numTri];

	// initialize integrator

	// find space needed for results, one slot per gpu worker
	long long maxDivisions = (int) (maxLength/ds + 1); // max num samples per side, rounded up
	// maximum possible number of samples per triangle is loosely upper bounded by 2 * maxDivisions^2
	// assumming edge lengths are bounded above by maxDivisions * 2
	long long resultSlots = max(2 * maxDivisions * maxDivisions, (long long) maxX * maxY); // at least num pixels
	integrator.initialize(pixArr, triArr, maxX, maxY, APPROXTYPE, resultSlots);

	// create an initial approximation based on this triangulation
	updateApprox();
}

void ConstantApprox::initialize(int pixelRate) {
	// create points
	int numX = ceil(((double) maxX) / pixelRate) + 1; // number of samples in x direction
	int numY = ceil(((double) maxY) / pixelRate) + 1;
	double dx = ((double) maxX) / (numX - 1); // step size in x direction, remembering to get both endpoints
	double dy = ((double) maxY) / (numY - 1);

	// create shared space for points
	numPoints = numX * numY;
	hipMallocManaged(&points, numPoints * sizeof(Point));

	for(int i = 0; i < numX; i++) {
		bool isBoundX = (i == 0) || (i == numX - 1); // whether point is on vertical boundary
		for(int j = 0; j < numY; j++) {
			bool isBoundY = (j == 0) || (j == numY - 1);
			int index1D = i * numY + j;
			// shift by (-0.5, -0.5) to align to edge of image (lattice points at pixel centers)
			points[index1D] = Point(i * dx - 0.5, j * dy - 0.5, isBoundX, isBoundY);
		}
	}
	cout << "starting grid: " << numX << "x" << numY << endl;

	// create triangles
	numTri = 2 * (numX - 1) * (numY - 1);
	hipMallocManaged(&triArr, numTri * sizeof(Triangle));
	hipMallocManaged(&grays, numTri * sizeof(double));
	imageInt = new double[numTri];

	int triInd = 0; // index the triangles
	for(int i = 0; i < numX; i++) {
		for(int j = 0; j < numY; j++) {
			int index1D = i * numY + j;
			// randomly triangulate the square with min x,y corner at this point
			if(i < numX - 1 && j < numY - 1) {
				Point *pt = points + index1D; // easier reference to current point
				if(rand() % 2 == 0) {
					triArr[triInd] = Triangle(pt, pt + numY, pt + numY + 1);
					faces.push_back({index1D, index1D + numY, index1D + numY + 1});
					triArr[triInd+1] = Triangle(pt, pt + numY + 1, pt + 1);
					faces.push_back({index1D, index1D + numY + 1, index1D + 1});
				} else {
					triArr[triInd] = Triangle(pt, pt + 1, pt + numY);
					faces.push_back({index1D, index1D + 1, index1D + numY});
					triArr[triInd+1] = Triangle(pt + numY, pt + 1, pt + numY + 1);
					faces.push_back({index1D + numY, index1D + 1, index1D + numY + 1});
				}
				triInd += 2;
			}
		}
	}

	double maxLength = 2 * max(dx, dy); // generously round up maximum triangle side length

	// initialize integrator

	// find space needed for results, one slot per gpu worker
	long long maxDivisions = (int) (maxLength/ds + 1); // max num samples per side, rounded up
	// maximum possible number of samples per triangle is loosely upper bounded by 2 * maxDivisions^2
	// assumming edge lengths are bounded above by maxDivisions * 2
	long long resultSlots = max(2 * maxDivisions * maxDivisions, (long long) maxX * maxY); // at least num pixels
	integrator.initialize(pixArr, triArr, maxX, maxY, APPROXTYPE, resultSlots);

	// create an initial approximation based on this triangulation
	updateApprox();
}

ConstantApprox::~ConstantApprox() {
	hipFree(pixArr);
	hipFree(points);
	hipFree(triArr);
	hipFree(grays);
	/*
	hipFree(reds);
	hipFree(greens);
	hipFree(blues);
	*/
	delete[] imageInt;
}

double ConstantApprox::computeEnergy() {
	return integrator.constantEnergyEval(grays, numTri, ds);
}

void ConstantApprox::computeGrad() {
	// clear gradients from last iteration
	for(int i = 0; i < numPoints; i++) {
		gradX[points + i] = 0;
		gradY[points + i] = 0;
	}
	for(int i = 0; i < numTri; i++) {
		// integral of fdA, retrieved from last updateApprox iteration
		double imageIntegral = imageInt[i];
		for(int j = 0; j < 3; j++) {
			double changeX, changeY;
			gradient(i, j, imageIntegral, &changeX, &changeY);
			// constrain points on boundary of image
			if(triArr[i].vertices[j]->isBorderX()) {
				changeX = 0;
			}
			if(triArr[i].vertices[j]->isBorderY()) {
				changeY = 0;
			}
			gradX[triArr[i].vertices[j]] += changeX;
			gradY[triArr[i].vertices[j]] += changeY;
		}
	}
}

void ConstantApprox::gradient(int t, int movingPt, double imageIntegral, double *gradX, double *gradY) {
	// to save time, only compute integrals if triangle is non-degenerate;
	// degenerate triangle has 0 energy and is locally optimal, set gradient to 0
	double area = triArr[t].getArea();
	double gradient[2] = {0, 0};
	if (area > TOLERANCE) {
		double dA[2] = {triArr[t].gradX(movingPt), triArr[t].gradY(movingPt)};
		double boundaryChange[2];
		// compute gradient in x and y direction
		for(int i = 0; i < 2; i++) {
			// sample more frequently because both time and space allow (or don't)
			boundaryChange[i] = integrator.lineIntEval(t, movingPt, (i == 0), ds);
		}
		for(int j = 0; j < 2; j++) {
			gradient[j] = (2 * area * imageIntegral * boundaryChange[j]
				- imageIntegral * imageIntegral * dA[j]) / (-area * area);
		}
	}
	// check for null pointers
	if (gradX && gradY) {
		*gradX = gradient[0];
		*gradY = gradient[1];
	}
}

bool ConstantApprox::gradUpdate() {
	// gradient descent update for each point
	for(int i = 0; i < numPoints; i++) {
		points[i].move(-stepSize * gradX.at(points+i), -stepSize * gradY.at(points+i));
	}
	// check validity of result
	for(int i = 0; i < numTri; i++) {
		if(triArr[i].getSignedArea() < 0) {
			return false;
		}
	}
	return true;
}

void ConstantApprox::undo() {
	for(int i = 0; i < numPoints; i++) {
		points[i].move(stepSize * gradX.at(points+i), stepSize * gradY.at(points+i));
	}
	stepSize /= 2;
}

void ConstantApprox::updateApprox() {
	for(int t = 0; t < numTri; t++) {
		// compute image dA and store it for reference on next iteration
		double val = integrator.doubleIntEval(t, ds);
		imageInt[t] = val;
		double area = triArr[t].getArea();
		// take average value
		double approxVal = val / area;
		// handle degeneracy
		if (isnan(approxVal)) {
			assert(area < TOLERANCE);
			approxVal = 255; // TODO: something better than this
		}
		grays[t] = min(255.0, approxVal); // prevent blowup in case of poor approximation
	}
}

double ConstantApprox::step(double &prevEnergy, double &newEnergy) {
	double usedStep;
	computeGrad();
    while(!gradUpdate()) {
        undo(); // keep halving stepSize until no triangle is inverted
	}
	updateApprox();
    prevEnergy = newEnergy;
	newEnergy = computeEnergy();
    // TODO: tune this
	if(newEnergy > prevEnergy) { // overshot optimum?
        do {
            undo();
        } while (!gradUpdate());
        updateApprox();
		newEnergy = computeEnergy();
		usedStep = stepSize;
    } else {
		usedStep = stepSize;
		stepSize *= 2; // prevent complete vanishing to zero
	}
    cout << "new energy: " << newEnergy << endl;
	cout << "Step size: " << usedStep << endl;
	return usedStep;
}

void ConstantApprox::run(int maxIter, double eps) {
	// track change in energy for stopping point
	double newEnergy = computeEnergy();
	// initialize to something higher than newEnergy
	double prevEnergy = newEnergy + 100 * eps;
	int iterCount = 0;
	while(iterCount < maxIter && abs(prevEnergy - newEnergy) > eps) {
		cout << "iteration " << iterCount << endl;
		step(prevEnergy, newEnergy);
		iterCount++;
	}
}

double ConstantApprox::getStep() {
	return stepSize;
}

// inefficient TODO: fix
vector<Point> ConstantApprox::getVertices() {
	vector<Point> vertices;
	for(int i = 0; i < numPoints; i++) {
		vertices.push_back(points[i]);
	}
	return vertices;
}

vector<array<int, 3>> ConstantApprox::getFaces() {
	return faces;
}

vector<array<double,3>> ConstantApprox::getColors() {
	vector<array<double, 3>> fullColors;
	for(int t = 0; t < numTri; t++) {
		// scale to fit polyscope colors TODO: check that this is correct
		int scale = 255;
		double area = triArr[t].getArea();
		double r = integrator.doubleIntEval(t, ds, RED) / (scale * area);
		double g = integrator.doubleIntEval(t, ds, GREEN) / (scale * area);
		double b = integrator.doubleIntEval(t, ds, BLUE) / (scale * area);
		fullColors.push_back({r, g, b});
	}
	return fullColors;
}
