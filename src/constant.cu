#include "hip/hip_runtime.h"
#include "constant.cuh"

const double TOLERANCE = 1e-10;

// custom rounding function to support needed pixel rounding
int customRound(double x) {
	int floor = (int) x;
	if (abs(x - floor) <= 0.5) {
		return floor;
	} else if (x > 0) {
		return floor + 1;
	}
	return floor - 1;
}

ConstantApprox::ConstantApprox(CImg<unsigned char> *img, vector<Point> *pts, vector<array<int, 3>> &inds, double step, double ds_) 
: stepSize(step), ds(ds_) {
	// create pixel array representation
	maxX = img->width();
	maxY = img->height();
	cout << "image is " << maxX << "x" << maxY << endl;
	// allocate shared space for pixel array
	hipMallocManaged(&pixArr, maxX * maxY * sizeof(Pixel));
	// create space for results, one slot per gpu worker
	int maxDivisions = (int) (max(maxX, maxY)/ds + 1); // max num samples per image side, rounded up
	// maximum possible number of samples per triangle is loosely upper bounded by 2 * maxDivisions^2
	hipMallocManaged(&results, 2 * maxDivisions * maxDivisions * sizeof(double));
	bool isGrayscale = (img->spectrum() == 1);
	for(int x = 0; x < maxX; x++) {
		for(int y = 0; y < maxY; y++) {
			int ind = x * maxY + y; // 1D pixel index
			if(isGrayscale) {
				pixArr[ind] = Pixel(x, y, (*img)(x, y));
			} else {
				int rgb[3];
				for(int i = 0; i < 3; i++) {
					rgb[i] = (*img)(x, y, 0, i);
				}
				int r = (*img)(x, y, 0, 0);
				pixArr[ind] = Pixel(x, y, rgb[0], rgb[1], rgb[2]);
			}
		}
	}

	// load in points of triangulation
	numPoints = pts->size();
	// allocate shared space for points
	hipMallocManaged(&points, numPoints * sizeof(Point));
	// copy everything in TODO: make this more efficient (get directly from source)
	for(int i = 0; i < numPoints; i++) {
		points[i] = pts->at(i);
	}

	// now load in all the triangles
	numTri = inds.size();
	// allocate shared space for triangles and colors
	hipMallocManaged(&triArr, numTri * sizeof(Triangle));
	hipMallocManaged(&grays, numTri * sizeof(double));
	/*
	hipMallocManaged(&reds, numTri * sizeof(double));
	hipMallocManaged(&greens, numTri * sizeof(double));
	hipMallocManaged(&blues, numTri * sizeof(double));
	*/
	for(int i = 0; i < numTri; i++) {
		array<int, 3> t = inds.at(i); // vertex indices for this triangle
		// constructor takes point addresses
		triArr[i] = Triangle(points + t.at(0), points + t.at(1), points + t.at(2));
	}
	imageInt = new double[numTri];
	// create shared space for triangle iterations
	hipMallocManaged(&workingTriangle, 3 * sizeof(Point));

	// create an initial approximation based on this triangulation
	updateApprox();
}

ConstantApprox::~ConstantApprox() {
	hipFree(pixArr);
	hipFree(results);
	hipFree(points);
	hipFree(triArr);
	hipFree(grays);
	/*
	hipFree(reds);
	hipFree(greens);
	hipFree(blues);
	*/
	hipFree(workingTriangle);
	delete[] imageInt;
}

double ConstantApprox::computeEnergy() {
	return constantEnergyEval(pixArr, maxX, maxY, triArr, grays, numTri, results);
}

void ConstantApprox::computeGrad() {
	// clear gradients from last iteration
	for(int i = 0; i < numPoints; i++) {
		gradX[points + i] = 0;
		gradY[points + i] = 0;
	}
	for(int i = 0; i < numTri; i++) {
		// integral of fdA, retrieved from last updateApprox iteration
		double imageIntegral = imageInt[i];
		for(int j = 0; j < 3; j++) {
			double changeX, changeY;
			gradient(i, j, imageIntegral, &changeX, &changeY);
			// constrain points on boundary of image
			if(triArr[i].vertices[j]->isBorderX()) {
				changeX = 0;
			}
			if(triArr[i].vertices[j]->isBorderY()) {
				changeY = 0;
			}
			gradX[triArr[i].vertices[j]] += changeX;
			gradY[triArr[i].vertices[j]] += changeY;
		}
	}
}

void ConstantApprox::gradient(int t, int movingPt, double imageIntegral, double *gradX, double *gradY) {
	// to save time, only compute integrals if triangle is non-degenerate;
	// degenerate triangle has 0 energy and is locally optimal, set gradient to 0
	double area = triArr[t].getArea();
	double gradient[2] = {0, 0};
	if (area > TOLERANCE) {
		double dA[2] = {triArr[t].gradX(movingPt), triArr[t].gradY(movingPt)};
		double boundaryChange[2];
		// compute gradient in x direction
		boundaryChange[0] = lineIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, movingPt, true, results, workingTriangle);
		// and in y direction
		boundaryChange[1] = lineIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, movingPt, false, results, workingTriangle);
		for(int j = 0; j < 2; j++) {
			gradient[j] = (2 * area * imageIntegral * boundaryChange[j]
				- imageIntegral * imageIntegral * dA[j]) / (-area * area);
		}
	}
	// check for null pointers
	if (gradX && gradY) {
		*gradX = gradient[0];
		*gradY = gradient[1];
	}
}

bool ConstantApprox::gradUpdate() {
	// gradient descent update for each point
	for(int i = 0; i < numPoints; i++) {
		points[i].move(-stepSize * gradX.at(points+i), -stepSize * gradY.at(points+i));
	}
	// check validity of result
	for(int i = 0; i < numTri; i++) {
		if(triArr[i].getSignedArea() < 0) {
			return false;
		}
	}
	return true;
}

void ConstantApprox::undo() {
	for(int i = 0; i < numPoints; i++) {
		points[i].move(stepSize * gradX.at(points+i), stepSize * gradY.at(points+i));
	}
	stepSize /= 2;
}

void ConstantApprox::updateApprox() {
	for(int t = 0; t < numTri; t++) {
		// compute image dA and store it for reference on next iteration
		double val = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results);
		imageInt[t] = val;
		double area = triArr[t].getArea();
		// take average value
		double approxVal = val / area;
		// handle degeneracy
		if (isnan(approxVal)) {
			assert(area < TOLERANCE);
			approxVal = 255; // TODO: something better than this
		}
		grays[t] = approxVal;
		// get rgb values
		/*
		reds[t] = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, red) / area;
		greens[t] = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, green) / area;
		blues[t] = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, blue) / area;
		*/
	}
}

double ConstantApprox::step(double &prevEnergy, double &newEnergy) {
	double usedStep;
	computeGrad();
    while(!gradUpdate()) {
        undo(); // keep halving stepSize until no triangle is inverted
	}
	updateApprox();
    prevEnergy = newEnergy;
	newEnergy = computeEnergy();
    // TODO: tune this
	if(newEnergy > prevEnergy) { // overshot optimum?
        do {
            undo();
        } while (!gradUpdate());
        updateApprox();
		newEnergy = computeEnergy();
		usedStep = stepSize;
    } else {
		usedStep = stepSize;
		stepSize *= 2; // prevent complete vanishing to zero
	}
    cout << "new energy: " << newEnergy << endl;
	cout << "Step size: " << usedStep << endl;
	return usedStep;
}

void ConstantApprox::run(int maxIter, double eps) {
	// track change in energy for stopping point
	double newEnergy = computeEnergy();
	// initialize to something higher than newEnergy
	double prevEnergy = newEnergy + 100 * eps;
	int iterCount = 0;
	while(iterCount < maxIter && abs(prevEnergy - newEnergy) > eps) {
		cout << "iteration " << iterCount << endl;
		step(prevEnergy, newEnergy);
		iterCount++;
	}
}

double ConstantApprox::getStep() {
	return stepSize;
}

// inefficient TODO: fix
vector<Point> ConstantApprox::getVertices() {
	vector<Point> vertices;
	for(int i = 0; i < numPoints; i++) {
		vertices.push_back(points[i]);
	}
	return vertices;
}

vector<array<double,3>> ConstantApprox::getColors() {
	vector<array<double, 3>> fullColors;
	for(int t = 0; t < numTri; t++) {
		// scale to fit polyscope colors TODO: check that this is correct
		int scale = 255;
		int area = triArr[t].getArea();
		double r = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, RED)/(scale * area);
		double g = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, GREEN)/(scale * area);
		double b = doubleIntEval(APPROXTYPE, pixArr, maxX, maxY, triArr, t, results, BLUE)/(scale * area);
		fullColors.push_back({r, g, b});
	}
	return fullColors;
}
