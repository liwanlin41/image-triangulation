#include "hip/hip_runtime.h"
#include "parallelInt.cuh"

// thread setup
int numThreadsX = 32;
int numThreadsY = 16;
dim3 threadsPerBlock(numThreadsX, numThreadsY);
// NOTE: block grid will have to be set within each function

// compute the sum of an array arr with given size, in parallel
// with 1D thread/blocks, storing the result per block in result
__global__ void sumBlock(double *arr, int size, double *result) {
	extern __shared__ double partial[]; // hold partial results
	int tid = threadIdx.x;
	int ind = blockIdx.x * blockDim.x + tid;
	// load into partial result array
	if(ind < size) {
		partial[tid] = arr[ind];
	} else {
		partial[tid] = 0;
	}
	__syncthreads();

	for(int step = blockDim.x / 2; step > 0; step /= 2) {
		if(tid < step) {
			partial[tid] += partial[tid + step];
		}
		__syncthreads();
	}

	// write output for block to result
	if(tid == 0) {
		result[blockIdx.x] = partial[0];
	}
}

// quickly sum an array with given size in parallel and return the result;
// NOTE: arr, partialRes must already be shared between host and device 
double sumArray(double *arr, int size, double *partialRes) {
	int numThreads = 1024; // threads per block
	// shared memory size for device
	int memSize = numThreads * sizeof(double);
	int numBlocks = (size + numThreads - 1) / numThreads;
	sumBlock<<<numBlocks, numThreads, memSize>>>(arr, size, partialRes);
	// number of elements to sum is now numBlocks
	// number of blocks for next iteration
	int newNumBlocks = (numBlocks + numThreads - 1) / numThreads;
	// repeat until all elements have been summed
	while(numBlocks > 1) {
		sumBlock<<<newNumBlocks,numThreads, memSize>>>(partialRes, numBlocks, partialRes); 
		numBlocks = newNumBlocks;
		newNumBlocks = (newNumBlocks + numThreads - 1) / numThreads;
	}
	// at this point the array has been summed and the result is in partialRes[0]
	hipDeviceSynchronize();
	double output = partialRes[0];
	return output;
}

// compute double integral of f dA for a single pixel and single triangle triArr[t]
// pixArr is a 1D representation of image, where pixel (x, y) is at x * maxY + y
// reults holds the result for each pixel
__global__ void pixConstantDoubleInt(Pixel *pixArr, int maxX, int maxY, Triangle *triArr, int t, double *results, ColorChannel channel) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr
	if(x < maxX && y < maxY) { // check bounds
		//double area = pixArr[ind].intersectionArea(triArr[t]);
		double area = pixArr[ind].approxArea(triArr[t]);
		results[ind] = area * pixArr[ind].getColor(channel);
	}
}

double doubleIntEval(ApproxType approx, Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, int &t, double *results, ColorChannel channel) {
	dim3 numBlocks((maxX + numThreadsX -1) / numThreadsX, (maxY + numThreadsY -1) / numThreadsY);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant: {
			pixConstantDoubleInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, triArr, t, results, channel);
			break;
		}
		case linear: // TODO: fill out
			break;
		case quadratic: // TODO: fill out
			break;
	}
	double answer = sumArray(results, maxX * maxY, results);
	hipDeviceSynchronize(); // wait for everything to finish
	return answer;
}

// compute the energy of a single pixel on triangle triArr[t]
__global__ void pixConstantEnergyInt(Pixel *pixArr, int maxX, int maxY, Triangle *triArr, double *colors, int t, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr;
	if(x < maxX && y < maxY) {
		//double area = pixArr[ind].intersectionArea(triArr[t]);
		double area = pixArr[ind].approxArea(triArr[t]);
		double diff = colors[t] - pixArr[ind].getColor();
		results[ind] = diff * diff * area;
	}
}

// using Point a as vertex point, sample ~samples^2/2 points inside the triangle with an area element of dA
// NOTE: samples does not count endpoints along edge bc as the parallelograms rooted there lie outside the triangle
// maxY is for converting 2D pixel index to 1D index
__global__ void approxConstantEnergySample(Pixel *pixArr, int maxY, Point *a, Point *b, Point *c, double color, double *results, double dA, int samples) {
	int u = blockIdx.x * blockDim.x + threadIdx.x; // component towards b
	int v = blockIdx.y * blockDim.y + threadIdx.y; // component towards c
	int ind = (2 * samples - u + 1) * u / 2 + v; // 1D index in results
	// this is because there are s points in the first column, s-1 in the next, etc. up to s - u + 1
	if(u + v < samples) {
		// get coordinates of this point using appropriate weights
		double x = (a->getX() * (samples - u - v) + b->getX() * u + c->getX() * v) / samples;
		double y = (a->getY() * (samples - u - v) + b->getY() * u + c->getY() * v) / samples;
		// find containing pixel
		int pixX = pixelRound(x);
		int pixY = pixelRound(y);
		double diff = color - pixArr[pixX * maxY + pixY].getColor();
		if(u + v == samples - 1) { // along the opposite edge, dA element is a triangle instead of a parallelogram
			results[ind] = diff * diff * dA / 2;
		} else {
			results[ind] = diff * diff * dA;
		}
	}
}

double constantEnergyApprox(Pixel *pixArr, int &maxY, Triangle *triArr, double *colors, int &numTri, double *results, double ds, Point *workingTri) {
	double totalEnergy = 0;
	for(int t = 0; t < numTri; t++) {
		// compute number of samples needed
		double maxLength = triArr[t].maxLength();
		int samples = ceil(maxLength/ds);
		int i = triArr[t].minVertex(); // vertex opposite shortest side
		// ensure minVertex is copied into location workingTri
		triArr[t].copyVertices(workingTri+((3-i)%3), workingTri+((4-i)%3), workingTri+((5-i)%3));
		// unfortunately half of these threads will not be doing useful work; fix this somehow?
		dim3 numBlocks((samples + numThreadsX - 1) / numThreadsX, (samples + numThreadsY - 1) / numThreadsY);
		double dA = triArr[t].getArea() * 2 / (samples * samples);
		approxConstantEnergySample<<<numBlocks, threadsPerBlock>>>(pixArr, maxY, workingTri, workingTri + 1, workingTri + 2, colors[t], results, dA, samples);
		totalEnergy += sumArray(results, samples * (samples + 1) / 2, results);
	}
	return totalEnergy;
}

double constantEnergyEval(Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, double *colors, int &numTri, double *results) {
	dim3 numBlocks((maxX + numThreadsX - 1) / numThreadsX, (maxY + numThreadsY - 1) / numThreadsY);
	double totalEnergy = 0;
	for(int t = 0; t < numTri; t++) {
		pixConstantEnergyInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, triArr, colors, t, results);
		totalEnergy += sumArray(results, maxX * maxY, results); // add energy for this triangle
	}
	return totalEnergy;
}

// compute line integral of v dot n f ds for a single pixel and single triangle a, b, c when point b is moving
__global__ void pixConstantLineInt(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, Point *c, bool isX, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y;
	if (x < maxX && y < maxY) {
		double answer = 0;
		for(int i = 0; i < 2; i++) { // v dot n is nonzero only on a -- b and b -- c
			// extract segment and maintain ccw order for outward normal
			Segment seg = (i == 0) ? Segment(a, b) : Segment(b, c);
			Point *segEnd = (i == 0) ? a : c; // determine endpoint of seg that is not b
			double midX, midY; // to hold midpoint of segment intersection with this pixel
			double length = pixArr[ind].intersectionLength(seg, &midX, &midY);
			if(length != 0) {
				Point midpoint(midX, midY);
				// compute velocity at this point by scaling
				double distanceToVertex = midpoint.distance(*segEnd);
				double scale = distanceToVertex / seg.length(); // 1 if at b, 0 at opposite edge
				// velocity components
				double velX = (isX) ? scale : 0;
				double velY = scale - velX;
				// get unit normal values for this segment
				double nx, ny;
				seg.unitNormal(&nx, &ny);
				double vn = velX * nx + velY * ny; // average value of v * n
				answer += vn * length * pixArr[ind].getColor();
			}
		}
		results[ind] = answer;
	}
}

double lineIntEval(ApproxType approx, Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, int &t, int &pt, bool isX, double *results, Point *workingTri) {
	dim3 numBlocks((maxX + numThreadsX - 1) / numThreadsX, (maxY + numThreadsY - 1) / numThreadsY);
	triArr[t].copyVertices(workingTri, workingTri+1, workingTri+2);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant: {
			pixConstantLineInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, workingTri+((pt+2)%3), workingTri+pt, workingTri+((pt+1)%3), isX, results);
			break;
		}
		case linear: // TODO
			break;
		case quadratic: // TODO
			break;
	}
	double answer = sumArray(results, maxX * maxY, results);
	return answer;
}