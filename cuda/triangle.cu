#include "hip/hip_runtime.h"
#include "triangle.cuh"

// custom rounding function to support needed pixel rounding

Triangle::Triangle(Point *a, Point *b, Point *c) {
	vertices[0] = a;
	vertices[1] = b;
	vertices[2] = c;
	if(getSignedArea() < 0) { // reverse direction
		vertices[1] = c;
		vertices[2] = b;
	}
}

double Triangle::getSignedArea() {
	double ax = vertices[0]->getX();
	double ay = vertices[0]->getY();
	double bx = vertices[1]->getX();
	double by = vertices[1]->getY();
	double cx = vertices[2]->getX();
	double cy = vertices[2]->getY();
	Matrix matrix(bx - ax, cx - ax, by - ay, cy - ay);
	return matrix.determinant()/2;
}

double Triangle::getArea() {
	double signedArea = getSignedArea();
	if (signedArea < 0) {
		return -signedArea;
	}
	return signedArea;
}

double Triangle::dA(int &p, double vx, double vy) {
	// first extract the other two endpoints; note order matters
	Point* edgePoints[2];
	// retrieve in ccw order
	edgePoints[0] = vertices[(p+1)%3];
	edgePoints[1] = vertices[(p+2)%3];
	// change is -velocity dot edge normal of length |e|/2
	Segment opposite(edgePoints[0], edgePoints[1]);
	Matrix velocity(vx, vy);
	Matrix norm = opposite.scaledNormal();
	// 1 by 1
	Matrix grad = velocity.transpose().multiply(norm);
	return -grad.get(0,0);
}

double Triangle::gradX(int &p) {
	return dA(p, 1, 0);
}

double Triangle::gradY(int &p) {
	return dA(p, 0, 1);
}

__device__ bool Triangle::contains(Point &p) {
	// p is inside the triangle iff the orientations of the triangles
	// with vertices (vertices[i], vertices[i+1], p) are all the same
	bool signs[3]; // hold signs of triangles (true if ccw)
	for(int i = 0; i < 3; i++) {
		signs[i] = (Triangle::getSignedArea(vertices[i], vertices[(i+1)%3], &p) >= 0);
	}
	return signs[0] == signs[1] && signs[1] == signs[2];
}

void Triangle::copyVertices(Point *ptrA, Point *ptrB, Point *ptrC) {
	*ptrA = *vertices[0];
	*ptrB = *vertices[1];
	*ptrC = *vertices[2];
}

double Triangle::getSignedArea(Point *a, Point *b, Point *c) {
	double ax = a->getX();
	double ay = a->getY();
	double bx = b->getX();
	double by = b->getY();
	double cx = c->getX();
	double cy = c->getY();
	Matrix matrix(bx - ax, cx - ax, by - ay, cy - ay);
	return matrix.determinant()/2;
}

ostream& operator<<(ostream& os, const Triangle &t) {
	os << "Triangle ";
	for(Point *ptr : t.vertices) {
		os << *ptr << " ";
	}
	os << "\n";
	return os;
}
