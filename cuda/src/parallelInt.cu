#include "hip/hip_runtime.h"
#include "parallelInt.cuh"

// thread setup
int numThreadsX = 32;
int numThreadsY = 32;
dim3 threadsPerBlock(numThreadsX, numThreadsY);
// NOTE: block grid will have to be set within each function

// compute the sum of an array arr with given size, in parallel
// with 1D thread/blocks, storing the result per block in result
__global__ void sumBlock(double *arr, int size, double *result) {
	extern __shared__ double partial[]; // hold partial results
	int tid = threadIdx.x;
	int ind = blockIdx.x * blockDim.x + tid;
	// load into partial result array
	if(ind < size) {
		partial[tid] = arr[ind];
	} else {
		partial[tid] = 0;
	}
	__syncthreads();

	for(int step = blockDim.x / 2; step > 0; step /= 2) {
		if(tid < step) {
			partial[tid] += partial[tid + step];
		}
		__syncthreads();
	}

	// write output for block to result
	if(tid == 0) {
		result[blockIdx.x] = partial[0];
	}
}

// quickly sum an array with given size in parallel and return the result;
// NOTE: arr, partialRes must already be shared between host and device 
double sumArray(double *arr, int size, double *partialRes) {
	int numThreads = 1024; // threads per block
	// shared memory size for device
	int memSize = numThreads * sizeof(double);
	int numBlocks = (size + numThreads - 1) / numThreads;
	sumBlock<<<numBlocks, numThreads, memSize>>>(arr, size, partialRes);
	// number of elements to sum is now numBlocks
	// number of blocks for next iteration
	int newNumBlocks = (numBlocks + numThreads - 1) / numThreads;
	// repeat until all elements have been summed
	while(numBlocks > 1) {
		sumBlock<<<newNumBlocks,numThreads, memSize>>>(partialRes, numBlocks, partialRes); 
		numBlocks = newNumBlocks;
		newNumBlocks = (newNumBlocks + numThreads - 1) / numThreads;
	}
	// at this point the array has been summed and the result is in partialRes[0]
	hipDeviceSynchronize();
	double output = partialRes[0];
	return output;
}

// compute double integral of f dA for a single pixel and single triangle triArr[t]
// pixArr is a 1D representation of image, where pixel (x, y) is at x * maxY + y
// reults holds the result for each pixel
__global__ void pixConstantDoubleInt(Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, int &t, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr
	if(ind < maxX * maxY) { // check bounds
		double area = pixArr[ind].intersectionArea(triArr[t]);
		results[ind] = area * pixArr[ind].getColor();
	}
}

double doubleIntEval(ApproxType approx, Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, int &t, double *results) {
	dim3 numBlocks((maxX + numThreadsX -1) / numThreadsX, (maxY + numThreadsY -1) / numThreadsY);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant:
			pixConstantDoubleInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, triArr, t, results);
			break;
		case linear: // TODO: fill out
			break;
		case quadratic: // TODO: fill out
			break;
	}
	double answer = sumArray(results, maxX * maxY, results);
	hipDeviceSynchronize(); // wait for everything to finish
	return answer;
}

// compute the energy of a single pixel on triangle triArr[t]
__global__ void pixConstantEnergyInt(Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, double *colors, int &t, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr;
	if(ind < maxX * maxY) {
		double area = pixArr[ind].intersectionArea(triArr[t]);
		double diff = colors[t] - pixArr[ind].getColor();
		results[ind] = diff * diff * area;
	}
}

double constantEnergyEval(Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, double *colors, int &numTri, double *results) {
	dim3 numBlocks((maxX + numThreadsX - 1) / numThreadsX, (maxY + numThreadsY - 1) / numThreadsY);
	double totalEnergy = 0;
	for(int t = 0; t < numTri; t++) {
		pixConstantEnergyInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, triArr, colors, t, results);
		totalEnergy += sumArray(results, maxX * maxY, results); // add energy for this triangle
	}
	hipDeviceSynchronize(); // wait to finish
	return totalEnergy;
}

// compute line integral of v dot n f ds for a single pixel and single triangle a, b, c when point b is moving
__global__ void pixConstantLineInt(Pixel *pixArr, int &maxX, int &maxY, Point *a, Point *b, Point *c, bool isX, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y;
	if (ind < maxX * maxY) {
		double answer = 0;
		for(int i = 0; i < 2; i++) { // v dot n is nonzero only on a -- b and b -- c
			// extract segment and maintain ccw order for outward normal
			Segment seg = (i == 0) ? Segment(a, b) : Segment(b, c);
			Point *segEnd = (i == 0) ? a : c; // determine endpoint of seg that is not b
			double midX, midY; // to hold midpoint of segment intersection with this pixel
			double length = pixArr[ind].intersectionLength(seg, &midX, &midY);
			if(length != 0) {
				Point midpoint(midX, midY);
				// compute velocity at this point by scaling
				double distanceToVertex = midpoint.distance(*segEnd);
				double scale = distanceToVertex / seg.length(); // 1 if at b, 0 at opposite edge
				double velX = (isX) ? scale : 0;
				Matrix v(velX, scale - velX); // velocity vector
				Matrix n = seg.unitNormal();
				double vn = v.transpose().multiply(n).get(0,0); // average value of v * n
				answer += vn * length * pixArr[ind].getColor();
			}
		}
		results[ind] = answer;
	}
}

double lineIntEval(ApproxType approx, Pixel *pixArr, int &maxX, int &maxY, Triangle *triArr, int &t, int &pt, bool isX, double *results) {
	dim3 numBlocks((maxX + numThreadsX - 1) / numThreadsX, (maxY + numThreadsY - 1) / numThreadsY);
	Point vertices[3]; // vertices of triArr[t]
	triArr[t].copyVertices(vertices, vertices+1, vertices+2);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant:
			pixConstantLineInt<<<numBlocks, threadsPerBlock>>>(pixArr, maxX, maxY, vertices + ((pt+2)%3), vertices + pt, vertices + ((pt+1)%3), isX, results);
			break;
		case linear: // TODO
			break;
		case quadratic: // TODO
			break;
	}
	double answer = sumArray(results, maxX * maxY, results);
	hipDeviceSynchronize();
	return answer;
}